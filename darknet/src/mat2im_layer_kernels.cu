#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "mat2im_layer.h"
extern "C" {
#include "hip/hip_runtime.h"
}


__global__ void mat2im_kernel(int index, unsigned char *in, int n, int w, int h, int c, unsigned char *out, float ratio, float *resizeout)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;
    int iid = id;
    int k = 2 - (id%c);
    id /= c;
    int j = id % w;
    id /= w;
    int i = id;
    int start = w * (index%4) + (index/4) * h * w * 4;
    int out_index = k*8*w*h + start + i*w*4 + j;
    //int in_index = (2-k)+c*j+c*w*i;
    out[out_index] = in[iid];

    i *= ratio;
    j *= ratio;
    w *= ratio;
    h *= ratio;
    start = w * (index%4) + (index/4) * h * w * 4;
    out_index = k*8*w*h + start + i*w*4 + j;
    resizeout[out_index] = in[iid] / 255.0;
}

void mat2im(int index, unsigned char *in, int n, int w, int h, int c, float *out, float ratio, float *resizeout)
{
    mat2im_kernel<<<cuda_gridsize(n), BLOCK>>>(index, in, n, w, h, c, (unsigned char*)out, ratio, resizeout);
    check_error(hipPeekAtLastError());
}
